#include "hip/hip_runtime.h"
//  diffusion_3D_RMS
//
//  Update Journal:
//  -- 03/11/2019: equal step length random leap, one fiber, IAS, 3D, cuda version
//  -- 04/25/2019: implement mitochondria, high permeability, short T2, same diffusivity as IAS
//  -- 01/29/2020: implement generalized realistic microstructure simulator (RMS): elastic reflection, water exchange (no permeability) and T2 relaxation
//  -- 07/13/2020: PGSE signal and thrust pointer
//  -- 01/10/2021: I/O .bin instead of .txt
//  -- 03/04/2021: get parameters from command line
//  -- 03/24/2021: create anisotropic free diffusivity
//  Created by Hong-Hsi Lee, Massachusetts General Hospital
//  Modified by Ricardo Coronado-Leija, New York University

#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <math.h>
#include <iomanip>
#include <time.h>
#include <cstdlib>
#include <algorithm>
#include <string>
#include <complex>
#include <unistd.h> // small
#include <getopt.h> // large
#include <cstdint>  // uint8_t

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

using namespace std;
    
#define Pi 3.14159265
// #define timepoints 1000
#define Ngrad_max 2000 
#define nite 4
#define Nc_max 3
#define DEB 0
#define Nbin 400

// ********** cuda kernel **********
__device__ double atomAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));
        
        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    
    return __longlong_as_double(old);
}

__global__ void setup_kernel(hiprandStatePhilox4_32_10_t *state, unsigned long seed){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}
__global__ void propagate(
    hiprandStatePhilox4_32_10_t *state, 
    double *sig0, 
    double *sig,   // pgse
    double *sigRe, // narrow pulse
    double *dx2, 
    double *dx4, 
    double *NPar_count,
    double *NPar_bin,
    const double *TD, 
    const int TN, 
    const double *stepDa, 
    const double *stepDr, 
    const double *T2, 
    const double *Pij, 
    const int Nc,  
    const int NPar, 
    const double res, 
    const double dt,
    const uint8_t *APix,
    const int NPix1, 
    const int NPix2, 
    const int NPix3, 
    const double *bvec,
    const double *bval, 
    const double *grad,
    const double *Delta,
    const double *delta,   
    const double *echotime,
    const int Ngrad,  
    const int timepoints,
    const bool pgse_flag,
    const int initFlag){

// cuda
int idx    = threadIdx.x + blockDim.x * blockIdx.x;
int stride = blockDim.x * gridDim.x;
hiprandStatePhilox4_32_10_t localstate=state[idx];    
// time step of the points that will be saved   
int Tstep  = TN/timepoints;

for(int k = idx; k < NPar; k += stride){

// ########################################################################################### //
    double step[Nc_max] = {0};
    for(int kk = 0; kk < Nc_max; kk++){
       step[kk] = max(stepDa[kk],stepDr[kk]);
       }

    #if DEB == 1
    for(int kk = 0; kk < Nc_max; kk++){
    printf("normalized axial  step size compartment %d = %.8f\n",kk,stepDa[kk]);
    printf("normalized radial step size compartment %d = %.8f\n",kk,stepDr[kk]);
    printf("normalized max    step size compartment %d = %.8f\n",kk,step[kk]);
    }  // kk  
    #endif

    // Random number
    double vRand = 0;    
    // Particle position on a grid
    int xParGi[3] = {0}, xParGj[3] = {0};
        
    // xi:   initial particle position
    // xt:   particle position at the i-th step
    // tt:   distance between particle and y-z, x-z, x-y box wall
    // vt:   a unit vector indicating the hopping direction
    // xTmp: a temporary variable to save the position
    double xi[3] = {0}, xt[3] = {0}, tt[3] = {0}, vt[3] = {0};
    // double xTmp[3] = {0};
        
    // for new direction    
    double cos_theta = 0, sin_theta = 0;
    int tidx = 0, nidx = 0;
     
    // size of the box
    int NPix[3] = {0};
    NPix[0] = NPix1; NPix[1] = NPix2; NPix[2] = NPix3;

    // Signal weighted by T2 relaxation
    double s0 = 0;

    // The flags of hitting the medium boundary
    bool flip0[3] = {false}, flip1[3] = {false};
    // bool flip0_tmp[3]={false}, flip1_tmp[3]={false};
    // bool flipx0=false, flipx1=false, flipy0=false, flipy1=false, flipz0=false, flipz1=false;

    // q = \gamma * g * \delta (1/µm)
    double qx = 0;                 // narrow pulse
    double phase = 0; // wide pulse
    double dx = 0, dy = 0, dz = 0;
    double x1[Ngrad_max*3] = {0};
    double pm[3] = {0}; pm[0] = 1; pm[1] = 1; pm[2] = 1;

    // fstep: remaining fraction of one step
    // tmp:   temporary variable
    // tmin:  the shortest distance between the particle and y-z, x-z, x-y plane (% of step)
    double fstep = 0, tmp = 0, tmin = 0; //, maxStep;

    // Elements of lookup table APix
    int ai = 0, aj = 0;
    unsigned long long int aidx;

    // The box wall hit by the particle. 1:y-z plane, 2:x-z plane, 3:x-y plane
    int ii_hit = 0;

    // The time staying in compartments
    double t[Nc_max] = {0};
//        printf("step size=%.4f\n",step[0]);
//        printf("step size=%.4f\n",step[1]);    
//        printf("before\n");
    // ********** Initialize Particle Positions inside compartments (Apix[index] > 0) ********** //
    while (1){
        if ( initFlag == 4 ) {
            xi[0] = 0.5*static_cast<double>(NPix1);
            xi[1] = 0.5*static_cast<double>(NPix2);
            xi[2] = 0.5*static_cast<double>(NPix3);
            break;
        } else {
        // random initial position
        xi[0] = hiprand_uniform_double(&localstate)*static_cast<double>(NPix1);
        xi[1] = hiprand_uniform_double(&localstate)*static_cast<double>(NPix2);
        xi[2] = hiprand_uniform_double(&localstate)*static_cast<double>(NPix3);
        
        // Whether the particle is inside compartments
        xParGi[0] = floor(xi[0]); 
        xParGi[1] = floor(xi[1]); 
        xParGi[2] = floor(xi[2]);
        aidx = (unsigned long long int)( (unsigned long long int) ( ((unsigned long long int)(NPix2) )*((unsigned long long int)(NPix1))*((unsigned long long int)(xParGi[2])) + ((unsigned long long int)(NPix1))*((unsigned long long int)(xParGi[1])) ) + (unsigned long long int)(xParGi[0]) );

        if ( initFlag == 3 ) {
            if ( APix[ (unsigned long long int)(aidx) ] !=0 ){ break; }
        } else {
            if ( APix[ (unsigned long long int)(aidx) ] == initFlag ){ break; } 
        }
        
        }
    } // while
    
    // printf("after\n");
    // ********** Simulate diffusion ********** //
    // updating particle position
    xt[0] = xi[0]; 
    xt[1] = xi[1]; 
    xt[2] = xi[2];
    // position of the particle on the voxelized geometry
    xParGi[0]=floor(xt[0]); 
    xParGi[1]=floor(xt[1]); 
    xParGi[2]=floor(xt[2]);
    // compartment
    aidx = (unsigned long long int)( (unsigned long long int) ( ((unsigned long long int)(NPix2) )*((unsigned long long int)(NPix1))*((unsigned long long int)(xParGi[2])) + ((unsigned long long int)(NPix1))*((unsigned long long int)(xParGi[1])) ) + (unsigned long long int)(xParGi[0]) );
    ai = (int)( APix[ (unsigned long long int)(aidx) ] );
        
    #if DEB == 1    
    printf("Particle Initial Position = %.2f, %.2f, %.2f\n",xt[0],xt[1],xt[2]);
    printf("Current Voxel = %d, %d, %d : %d\n",xParGi[0],xParGi[1],xParGi[2],ai);
    #endif 

    for(int i = 0; i < TN; i++){
    // ========================================================================================= //
    // ========================================================================================= //
    // ========================================================================================= //
        fstep = 1.0;
        // set boundary hitting flags to false (for over three directions)
        for(int jj = 0; jj < 3; jj++){
            flip0[jj] = false; 
            flip1[jj] = false;
            }
        // flipx0=false; flipx1=false; flipy0=false; flipy1=false; flipz0=false; flipz1=false;
        
        // Random elevation 
        vRand     = hiprand_uniform_double(&localstate);
        cos_theta = 1.0 - 2.0*vRand;
        sin_theta = 2.0*sqrt(vRand*(1.0 - vRand)); //sin(acos(cos_theta));
        // random azimuthal
        vRand = hiprand_uniform_double(&localstate);
        vt[0] = sin_theta*cos(2.0*Pi*vRand); // x
        vt[1] = sin_theta*sin(2.0*Pi*vRand); // y
        vt[2] = cos_theta;                   // z
       
        #if DEB == 1    
        printf("New direction = %.2f, %.2f, %.2f\n",vt[0],vt[1],vt[2]);
        #endif 

//         // rescaling new direction according to stepDa, stepDr
//         maxStep = max(stepDa[ai],stepDr[ai]);
//         vt[0]   = stepDr[ai]*vt[0]/maxStep;
//         vt[1]   = stepDr[ai]*vt[1]/maxStep;
//         vt[2]   = stepDa[ai]*vt[2]/maxStep;
// 
//         #if DEB == 1    
//         printf("Scaled direction = %.2f, %.2f, %.2f\n",vt[0],vt[1],vt[2]);
//         #endif 
     
            
        for(int j = 0; (j < nite) && (fstep > 0); j++){ // several attempts ? 
            
            // check hitting walls in the voxel    
            tmin   =  2.0; 
            ii_hit = -1.0;
            for(int ii = 0; ii < 3; ii++){
                if(vt[ii] > 0.0){ // positive movement   i       i+1
                    // distance particle to next voxel   |  *<--->|
                    tmp = static_cast<double>(xParGi[ii]) + 1.0 - xt[ii];
                    } 
                else if(vt[ii] < 0.0){ // negative movement      i       i+1 
                    // distance particle to current voxel (neg)  |<--->*  |         
                    tmp = static_cast<double>(xParGi[ii]) - xt[ii];
                    } 
                else{
                    tmp = 2.0; // vt[ii] == 0 ?
                    } // if-elseif-else     
            // tmp=static_cast<double>(xParGi[ii]) + fmax(0.0,static_cast<double>(vt[ii]>0.0)) - xt[ii];
                if(fabs(tmp) > fabs(vt[ii])){
                    // movement of the particle is still inside of the voxel
                    tt[ii] = 2.0;
                    } 
                else{
                    // movement could get particle outside of the voxel
                    tt[ii] = max(0.0,tmp/vt[ii]); // which percent (positive) of vt is tmp ?
                    } // if-else
                // particle crossed at least one voxel-wall (just save th shortest distance)
                // tmin is the percentage of the step needed to get to the respective wall    
                if(tt[ii] < tmin){
                    tmin   = tt[ii];
                    ii_hit = ii;
                    } // if

                #if DEB == 1    
                printf("ii = %d, vt = %.2f, tmp = %.2f, tt = %.2f, tmin = %.2f, ii_hit = %d \n",
                        ii,vt[ii],tmp,tt[ii],tmin,ii_hit);
                #endif  

                } // i
                
            // if ( ii_hit<0 ) {
            //    printf("Error1: walker does not encounter the box wall, tmin=%.4f, xGi=%i,%i,%i, xGj=%i,%i,%i\n",tmin,xParGi[0],xParGi[1],xParGi[2],xParGj[0],xParGj[1],xParGj[2]);
            //    break;
            // }
                
            // Should be impossible (tmin initial is zero, then it is assigned 2.0 or max(0,+num))
            if(tmin < 0.0){
                printf("Error: walker jumps into wrong direction, tmin = %.4f\n",tmin);
                break;
                }
            
            // Update Position 
            if( (fstep*step[ai]) >= tmin){ // just move until reaching the closest wall
                fstep = fstep - tmin/step[ai]; // remaining fraction of the step
                xt[0] = xt[0] + tmin*vt[0];
                xt[1] = xt[1] + tmin*vt[1];
                xt[2] = xt[2] + tmin*vt[2];
                t[ai] = t[ai] + tmin/step[ai];
                } 
            else{ // move the full step
                xt[0] = xt[0] + step[ai]*fstep*vt[0];
                xt[1] = xt[1] + step[ai]*fstep*vt[1];
                xt[2] = xt[2] + step[ai]*fstep*vt[2];
                t[ai] = t[ai] + fstep;

                #if DEB == 1    
                printf("Final Position = %.2f, %.2f, %.2f\n",xt[0],xt[1],xt[2]);
                #endif 

               // printf("Error3: walker does not encounter the box wall, tmin=%.4f, xGi=%i,%i,%i, xGj=%i,%i,%i, vt=%.4f, ii_hit=%i\n",tmin,xParGi[0],xParGi[1],xParGi[2],xParGj[0],xParGj[1],xParGj[2],vt[ii_hit],ii_hit);
                break;
                } // if-else
                
            // set position on the lookup table     
            xParGj[0] = xParGi[0]; 
            xParGj[1] = xParGi[1]; 
            xParGj[2] = xParGi[2];

            #if DEB == 1    
            printf("Position Closest Wall %d = %.2f, %.2f, %.2f : fstep=%f t=%f\n",i,xt[0],xt[1],xt[2],fstep,t[ai]);
            #endif 

           // if ( (~flipx0) && (~flipx1) && (~flipy0) && (~flipy1) && (~flipz0) && (~flipz1) ) {
           // if ( (~flip0[0]) && (~flip1[0]) && (~flip0[1]) && (~flip1[1]) && (~flip0[2]) && (~flip1[2]) ) {

            // I believe this goes here, otherwise we would be accessing a negative indice
            // but originally it was after the section on
            // behavior of the particle on the wall
            if ( ii_hit < 0 ){
                printf("Error: walker does not encounter the box wall, tmin=%.4f, xGi=%i,%i,%i, xGj=%i,%i,%i\n",tmin,xParGi[0],xParGi[1],xParGi[2],xParGj[0],xParGj[1],xParGj[2]);
                break;
                } // if

            // behavior of the particle on the wall
            if(vt[ii_hit] >= 0.0){ // positive movement
                xParGj[ii_hit] = xParGj[ii_hit] + 1; // go next voxel
                if(xParGj[ii_hit]  >= NPix[ii_hit]){ // if upper boundary of the medium reached 
                    xParGj[ii_hit]  = NPix[ii_hit] - 1;
                    flip1[ii_hit]   = true;          // mirror boundary conditions
                    vt[ii_hit]      = -vt[ii_hit];   // oposite direction 
                    continue;
                    } // if
                } // if 
            else{ // negative movement
                xParGj[ii_hit] = xParGj[ii_hit] - 1; // go previous voxel
                if(xParGj[ii_hit] < 0){              // if bottom boundary of the medium reached   
                    xParGj[ii_hit] = 0;
                    flip0[ii_hit]  = true;           // mirror boundary conditions
                    vt[ii_hit]     = -vt[ii_hit];    // oposite direction 
                    continue;
                    } // if
                } // else
                
            // get compartment at current voxel    
            aidx = (unsigned long long int)( (unsigned long long int) ( ((unsigned long long int)(NPix2) )*((unsigned long long int)(NPix1))*((unsigned long long int)(xParGj[2])) + ((unsigned long long int)(NPix1))*((unsigned long long int)(xParGj[1])) ) + (unsigned long long int)(xParGj[0]) );    

            aj = (int) ( APix[ (unsigned long long int)(aidx) ] );
               
            // Checking permeability
            if(Pij[ai*Nc + aj] > (1 - 1e-10) ){ // same compartment: full permeability
                ai = aj; // update voxel of hitted wall
                xParGi[ii_hit] = xParGj[ii_hit]; 
                } 
            else if(Pij[ai*Nc + aj] < 1e-10){ // different compartments: no permeability 
                vt[ii_hit] = -vt[ii_hit]; // change direction of diffusion
               // printf("ai=%i, aj=%i, xGj=%i,%i,%i\n",ai,aj,xParGj[0],xParGj[1],xParGj[2]);
               } 
            else{ // different compartment: some permeability
                // printf("ai=%i, aj=%i, Pij=%.4f\n",ai,aj,Pij[ai*Nc+aj]);
                vRand = hiprand_uniform_double(&localstate);
                if(vRand < Pij[ai*Nc+aj]){
                    ai = aj; // update voxel of hitted wall
                    xParGi[ii_hit] = xParGj[ii_hit];
                    } 
                else{
                    vt[ii_hit] = -vt[ii_hit]; // change direction of diffusion
                    }
                } // if-elseif-else

            #if DEB == 1    
            printf("New direction 2 = %.2f, %.2f, %.2f\n",vt[0],vt[1],vt[2]);
            printf("Current Voxel 2 = %d, %d, %d : %d\n",xParGi[0],xParGi[1],xParGi[2],ai);
            #endif 
                
            } // j - several attempts
            
        // Apply flipping due to mirroring boundary conditions (why outside j and xi ??? )
        for(int jj = 0; jj < 3; jj++){
            if(flip0[jj]){
                xi[jj] = -xi[jj];
                pm[jj] = -pm[jj];
                } // if
            if(flip1[jj]){
                xi[jj] = 2.0*static_cast<double>(NPix[jj])-xi[jj];
                pm[jj] = -pm[jj];
                } // if
            // reset flipping    
            flip0[jj] = false; 
            flip1[jj] = false;
            } // jj

        #if DEB == 1    
        printf("Current Position i %d = %.2f, %.2f, %.2f\n",i,xi[0],xi[1],xi[2]);
        printf("Current Position t %d = %.2f, %.2f, %.2f\n",i,xt[0],xt[1],xt[2]);
        #endif 

    if(pgse_flag){  // Gradient parameters of PGSE: Delta, delta, |g|, gx, gy, gz  (only saved if asked)

        // Displacement
        dx = (xt[0] - xi[0])*res*pm[0];
        dy = (xt[1] - xi[1])*res*pm[1];
        dz = (xt[2] - xi[2])*res*pm[2];

        // add phase 
        for(int j = 0; j < Ngrad; j++){
            // First Pulse
            if(static_cast<double>(i+1)*dt <= delta[j]){
                x1[3*j]+=dx; x1[3*j+1]+=dy; x1[3*j+2]+=dz;
            } // 1st
            else if ( (static_cast<double>(i+1)*dt > Delta[j]) & (static_cast<double>(i+1)*dt <= (Delta[j]+delta[j])) ){
                x1[3*j]-=dx; x1[3*j+1]-=dy; x1[3*j+2]-=dz;
            } // 2nd
        } // j

//         // add phase
//         for(int j = 0; j < Ngrad; j++){
//             // Second pulse
//             if( ( static_cast<double>(i + 1)*dt >= Delta[j] ) & ( static_cast<double>(i + 1)*dt < (Delta[j] + delta[j]) ) ){
//                 phase[j] -= grad[j] * (dx*bvec[j*3] + dy*bvec[j*3 + 1] + dz*bvec[j*3 + 2])*dt;
//                 } // 2nd 
//             } // j   
        
        // Readout (here it may be better to change t for echotime) and have one for each ngrad
        if ( i == (TN-1) ){ // just saves the last time
            s0 = 0.0;
            for(int j = 0; j < Nc; j++){
                s0 += (t[j]/T2[j]);
                } // j
            s0 = exp(-1.0*s0);
            // pgse signal
            for(int j = 0; j < Ngrad; j++){
                phase = grad[j] * ( x1[j*3]*bvec[j*3] + x1[j*3+1]*bvec[j*3+1] + x1[j*3+2]*bvec[j*3+2] )*dt;
                atomAdd(&sig[j],s0*cos(phase));
                } // j
        } // if
    } // pgse

    if ( (i%Tstep) == 0 ) { // Save moment tensor for dx^2 and dx^4, and signal for the b-table
        
        // T2 Relaxation
        s0 = 0.0;
        for(int j = 0; j < Nc; j++){
            s0 = s0 + (t[j]/T2[j]);
            } // j
        s0 = exp(-1.0*s0); // s0 = 1.0;
        
        tidx = i/Tstep;
        nidx = Nc*tidx + ai;
        
        atomAdd(&sig0[tidx],s0);       // Update s0 signal 
        atomAdd(&NPar_count[nidx],1);  // Update npar (per compartment) counter 
        
        if (initFlag==4){
        nidx = floor(sqrt((xt[0]-static_cast<double>(NPix1)/2.0)*(xt[0]-static_cast<double>(NPix1)/2.0) + (xt[1]-static_cast<double>(NPix2)/2.0)*(xt[1]-static_cast<double>(NPix2)/2.0) + (xt[2]-static_cast<double>(NPix3)/2.0)*(xt[2]-static_cast<double>(NPix3)/2.0))/(static_cast<double>(NPix1)/2.0/static_cast<double>(Nbin)));
        if (nidx<=Nbin){ atomAdd(&NPar_bin[Nbin*tidx+nidx],1); }
        }

        // Displacement
        dx = (xt[0] - xi[0])*res*pm[0];
        dy = (xt[1] - xi[1])*res*pm[1];
        dz = (xt[2] - xi[2])*res*pm[2];

        #if DEB == 1    
        printf("Current Position i %d = %.4f, %.4f, %.4f\n",i,xi[0]*res,xi[1]*res,xi[2]*res);
        printf("Current Position t %d = %.4f, %.4f, %.4f\n",i,xt[0]*res,xt[1]*res,xt[2]*res);
        printf("Current Displace t %d = %.4f, %.4f, %.4f\n",tidx,dx,dy,dz);
        #endif 
     
        // Second Order Moment
        atomAdd(&dx2[6*tidx+0],s0*dx*dx);
        atomAdd(&dx2[6*tidx+1],s0*dx*dy);
        atomAdd(&dx2[6*tidx+2],s0*dx*dz);
        atomAdd(&dx2[6*tidx+3],s0*dy*dy);
        atomAdd(&dx2[6*tidx+4],s0*dy*dz);
        atomAdd(&dx2[6*tidx+5],s0*dz*dz);
        
        // Fourth Order Moment
        atomAdd(&dx4[15*tidx+0] ,s0*dx*dx*dx*dx);
        atomAdd(&dx4[15*tidx+1] ,s0*dx*dx*dx*dy);
        atomAdd(&dx4[15*tidx+2] ,s0*dx*dx*dx*dz);
        atomAdd(&dx4[15*tidx+3] ,s0*dx*dx*dy*dy);
        atomAdd(&dx4[15*tidx+4] ,s0*dx*dx*dy*dz);
        atomAdd(&dx4[15*tidx+5] ,s0*dx*dx*dz*dz);
        atomAdd(&dx4[15*tidx+6] ,s0*dx*dy*dy*dy);
        atomAdd(&dx4[15*tidx+7] ,s0*dx*dy*dy*dz);
        atomAdd(&dx4[15*tidx+8] ,s0*dx*dy*dz*dz);
        atomAdd(&dx4[15*tidx+9] ,s0*dx*dz*dz*dz);
        atomAdd(&dx4[15*tidx+10],s0*dy*dy*dy*dy);
        atomAdd(&dx4[15*tidx+11],s0*dy*dy*dy*dz);
        atomAdd(&dx4[15*tidx+12],s0*dy*dy*dz*dz);
        atomAdd(&dx4[15*tidx+13],s0*dy*dz*dz*dz);
        atomAdd(&dx4[15*tidx+14],s0*dz*dz*dz*dz);
        
        // Diffusion Signal Narrow Pulse Limit (this will always be saved)
        // Acording to Rafael Patiño:
        // \delta Phase = a(t)*gamma*g(t)*z(t)*(\delta t)
        // Phase        = \sum_i \delta Phase_i
        // Signal       = \sum_i e^{ i * Phase _i}
        // Signal_{re}  = \sum_i cos( Phase_i )
        for(int j = 0; j < Ngrad; j++){ // bvalue = t*q^2 => q = sqrt( bvalue/t )
            qx = sqrt( bval[j] / TD[tidx] ) * (dx*bvec[j*3] + dy*bvec[j*3+1] + dz*bvec[j*3+2]);
            atomAdd(&sigRe[Ngrad*tidx+j] , s0*cos(qx));
            } // j

        } // if Tstep (save state)
    // ========================================================================================= //
    // ========================================================================================= //
    // ========================================================================================= //
    } // steps i 
// ########################################################################################### //    
} // particles k 
state[idx]=localstate;
} // function

// .......................................................................... //
// ########################################################################## //
// ########################################################################## //
// ########################################################################## //
// .......................................................................... //

int show_help(){
printf(
// === Non Optional Arguments === //
"rms [ options ] input btable output\n"
"\n"
"\tinput \n"
"\t\tname of the input (binary) file with the substrate/medium.\n"
"\tbtable\n"
"\t\tname of the btable (.txt) file to generate the diffusion signal.\n" 
"\t\tEach line of the file must be (with b in ms/um^2):\n"
"\t\t\t x1 y1 z1 b1\n"
"\t\t\t x2 y2 z2 b2\n"
"\t\t\t x3 y3 z3 b3\n"
"\t\t\t .  .  .  . \n"
"\t\t\t .  .  .  . \n"
"\t\t\t .  .  .  . \n"
"\t\t\t xn yn zn bn\n"
"\toutput\n"
"\t\tname of the output (binary) file with the results of the simulation.\n"     
"\n"
"Realistic Microstructure Simulator (RMS).\n"
"Gets D(t), K(t) and the time dependent diffusion signal for the given substrate.\n"
"\n"
// === Optional Arguments === //
"Options:\n"
"\n"
// === More Needed Optional Arguments === //
"\t -particles (p)\n"
"\t\tNumber of random walkers to simulate.\n" 
"\t\t(default: -particles 1e6).\n" 
"\t -time (t)\n"
"\t\ttotal time of the simulation, in ms.\n" 
"\t\t(default: -time 100).\n" 
"\t -dintra (i)\n"
"\t\tintra axonal space (IAS) diffusivity at time = 0, in um^2/ms.\n" 
"\t\t(default: -dintra 2).\n" 
"\t -dextra (e)\n"
"\t\textra axonal space (EAS) diffusivity at time = 0, in um^2/ms.\n" 
"\t\t(default: -dextra 2).\n" 
"\t -voxstep (v)\n"
"\t\tvoxstep takes values between 0 and 1.\n"
"\t\tthe length step is equal to voxstep * (voxel size).\n"    
"\t\tthe time step is then equal to (length step)^2/(6*D).\n"  
"\t\t(default: -voxstep 0.9).\n" 
"\t -mspoints (m)\n"
"\t\tNumber of points per ms to sample D(t), K(t) and diffusion signal.\n"
"\t\t(default: -mspoints 10).\n" 
"\t -space (s)\n"
"\t\tSelect in which space the simulation will be performed:\n"
"\t\t1 = IAS, 2 = EAS.\n"
"\t\t(default: -space 1).\n" 
"\t -pgse (g)\n"
"\t\tgenerates the diffusion signal for a PGSE sequence, saving it with the name specified, \n"
"\t\t\\Delta (D), \\delta (d) and TE should be indicated in the btable.\n"
"\t\tEach line of the file must be (with b in ms/um^2):\n"
"\t\t\t x1 y1 z1 b1 D1 d1 TE1\n"
"\t\t\t x2 y2 z2 b2 D2 d2 TE2\n"
"\t\t\t x3 y3 z3 b3 D3 d3 TE3\n"
"\t\t\t .  .  .  .  .  .   . \n"
"\t\t\t .  .  .  .  .  .   . \n"
"\t\t\t .  .  .  .  .  .   . \n"
"\t\t\t xn yn zn bn Dn dn TEn\n"
"\t -help (h)\n"
"\t\tshow this help\n"
"\n"
"Author:\n"
"Hong Hsi Lee (ORCID 0000-0002-3663-6559)\n"
"Ricardo Coronado-Leija\n"
"\n"
"References:\n"
"Lee, et al., Journal of Neuroscience Methods 2021 (doi:10.1016/j.jneumeth.2020.109018)\n"
"Fieremans & Lee., Neuroimage 2018 (doi:10.1016/j.neuroimage.2018.06.046)\n"
"\n");
return 0;
}    

int main(int argc, char *argv[]) {

// To check time
clock_t begin = clock();
clock_t end   = clock();

// Initialize seed for RNG
unsigned long seed = 0;
FILE *urandom;
urandom = fopen("/dev/random", "r");
fread(&seed,sizeof(seed),1,urandom);
fclose(urandom);

// .......................................................................... //
// ########################################################################## //
// .......................................................................... //

int option_index; // getopt_long_only stores the option index here.
int goloval;      // getopt_long_only returns the option value here

int    NPar       = 1000000;      // # particles
double dtime      = 100.0;        // total diffusion time 
double vstep      = 0.9;          // length_step = vstep*voxel_size;    
int    Nc         = 3;            // # compartments
double Dintra     = 2.0;          // ICS diffusion at time = 0 ms
double Dextra     = 2.0;          // ECS diffusion at time = 0 ms
double t2intra    = 1e20;         // ICS T2 time 
double t2extra    = 1e20;         // ECS T2 time
double kappa      = 0.0;          // permeability, um/ms
int    mspoints   = 10;           // points per mili-second
int    initFlag   = 3;            // 1 = ICS, 2 = ECS, 3 = ICS + ECS, 4 = center
bool   pgse_flag  = false;        // use PGSE sequence
bool   debug_flag = false;        // for debugging
bool   help_flag  = false;        // show help
char namePGSE[500];               // name output PGSE signal

// ======================== Reading Input Parameters ======================== //

// options
struct option long_options[] = {
// These options don’t set a flag. We distinguish them by their indices.
{"particles",       required_argument, 0, 'p'},
{"time",            required_argument, 0, 't'},
{"voxstep",         required_argument, 0, 'v'},
{"compartments",    required_argument, 0, 'c'},
{"dintra",          required_argument, 0, 'i'},
{"dextra",          required_argument, 0, 'e'},
{"t2intra",         required_argument, 0, 'r'},
{"t2extra",         required_argument, 0, 'a'},
{"permeability",    required_argument, 0, 'k'},
{"mspoints",        required_argument, 0, 'm'},
{"space",           required_argument, 0, 's'},
{"pgse",            required_argument, 0, 'g'},
{"debug",           no_argument      , 0, 'd'},
{"help",            no_argument      , 0, 'h'},
{0, 0, 0, 0}
};

while(true){
// Detecting the next option
goloval = getopt_long_only (argc, argv, "p:t:v:c:i:e:r:m:s:g:dh", long_options, &option_index);

// Detect the end of the options and break the while. 
if (goloval == -1) break;

switch (goloval){
  case 'p':
     NPar = atoi(optarg);
     break;
  case 't':
     dtime = atof(optarg);   
     break;
  case 'v':
     vstep = atof(optarg);   
     if(vstep < 0.0001 || 0.9999 < vstep){
     cout << "vstep = " << vstep << "is not a valid value,"
          << " it should have a value between 0 and 1, settting to 0.9" << endl;
     vstep = 0.9;   
     }
     break;
  case 'c':
     Nc = atoi(optarg);
     break;
  case 'i':
     Dintra = atof(optarg);     
  case 'e':
     Dextra = atof(optarg);          
     break;
  case 'r':
     t2intra = atof(optarg);     
     break;
  case 'a':
     t2extra = atof(optarg);     
     break;
  case 'k':
     kappa = atof(optarg);     
     break;
  case 'm':
     mspoints = atoi(optarg);
     if(mspoints < 1){
     cout << "mspoints = " << mspoints << "is not a valid value,"
          << " it should be an integer larger than 1, settting to 10" << endl; 
     mspoints = 10;
     }  
     break;
  case 's':
     initFlag = atoi(optarg);
     if(initFlag < 1 || initFlag > 4){ 
       cout << "Incorrect option, settting to IAS + EAS." << endl;
       initFlag = 1;
       } //
     break;     
  case 'g':
     pgse_flag = true;
     strcpy(namePGSE,optarg); 
     cout << "pgse_flag: " << pgse_flag << " optarg: " << namePGSE << endl;
     break;     
  case 'd':
     debug_flag = true;
     break;     
  case 'h':
     help_flag = true;
     break;
  case '?':
     //getopt_long already printed an error message.
     break;
  default:
     abort();
  } // switch
} // while (1)

// Showing the help
if(help_flag || argc == 1){
show_help();
exit(1);
}

// ======================================================================= //      
// Remaining command line arguments (not options).
if(argc - optind != 3){
printf("Expecting 3 arguments %d provided. Use option -help to show the help.\n",(argc - optind));
return 1;
}

// --- Non-Optional arguments --- //
char name_input[500], basename_output[500], name_btable[500];

sprintf(name_input     ,"%s",argv[optind++]);
sprintf(name_btable    ,"%s",argv[optind++]);
sprintf(basename_output,"%s",argv[optind++]);

// .......................................................................... //
// ########################################################################## //
// .......................................................................... //

// =========================== Load Mictostructure ========================== //

unsigned int i, j, k, Nbs, Nbvec, NPix1, NPix2, NPix3, vs;
double Nbvals, res, dl, dt, TN_TP; 
int TN, timetemp, timepoints;

// ===== Read b-table ===== //
cout << "Load b-table: " << name_btable << endl;
Nbs = 0; // counter elements in btable
ifstream myfile(name_btable,ios::in);
if(myfile.is_open()){
while(!myfile.eof()){
myfile >> Nbvals; Nbs++;
} // while
myfile.close();
} // if
else{
cout << "ERROR: Can't open file: " << name_btable << endl; 
exit(1); 
}
// decompose in columns (pgse 7 or narrow 4)
if(pgse_flag){
Nbvec = Nbs/7;
Nbs   = Nbvec*7;
if(Nbvec > Ngrad_max){
cout << "ERROR: Maxim number of ngrad should be: " << Ngrad_max << endl; 
} // if
} // pgse
else{
Nbvec = Nbs/4;
Nbs   = Nbvec*4;
} // narrow
// diffusion = mu m^2 / ms => bvalue = ms / mu m^2 => gyromagnetic ratio = 1 (s^(-1) T^(-1)) => 1/ (ms * mT)
// Saving btab directly on a host vector
// Narrow: [gx gy gz bval] 
// PGSE: gx, gy, gz, |g|, Delta, delta, TE 
thrust::host_vector<double> grad(Nbvec);   // gradient (mT/mu m)
thrust::host_vector<double> Delta(Nbvec);  // Delta    (ms)
thrust::host_vector<double> delta(Nbvec);  // delta    (ms)
thrust::host_vector<double> TE(Nbvec);     // TE       (ms)
thrust::host_vector<double> bval(Nbvec);   // bvector  (unitary)
thrust::host_vector<double> bvec(Nbvec*3); // bvalue   (ms / mu m^2)
ifstream myfile1(name_btable,ios::in);
// reading btable
if(pgse_flag){
cout << "Computing bvalues" << endl;
for(i = 0; i < Nbvec; i++) {
myfile1 >> bvec[i*3+0]; 
myfile1 >> bvec[i*3+1]; 
myfile1 >> bvec[i*3+2];
myfile1 >> bval[i];
myfile1 >> Delta[i];
myfile1 >> delta[i];
myfile1 >> TE[i]; 
grad[i] = sqrt(bval[i]/delta[i]/delta[i]/( Delta[i] - delta[i]/3.0 ));
} // i 
} // pgse
else{
for(i = 0; i < Nbvec; i++) {
myfile1 >> bvec[i*3+0]; 
myfile1 >> bvec[i*3+1]; 
myfile1 >> bvec[i*3+2];
myfile1 >> bval[i]; 
} // i  
} // narrow
myfile1.close();
// // debug
// for(i = 0; i < Nbvec; i++){
// cout << bvec[i*3+0] << " " << bvec[i*3+1] << " "  << bvec[i*3+2] << " " << bval[i] << endl;
// } // i 
cout << "N elements = " << Nbs << " Nbvec = " << Nbvec << endl;

// ===== Read medium substrate (.bin) ===== //
cout << "Reading Substrate: " << name_input << endl;
FILE *finput;   
finput = fopen(name_input, "rb");
if(!finput){
cout << "ERROR: Can't open file: " << name_input << endl; 
exit(1); 
} // if
fread(&NPix1,sizeof(unsigned int),1,finput); 
fread(&NPix2,sizeof(unsigned int),1,finput); 
fread(&NPix3,sizeof(unsigned int),1,finput); 
fread(&vs   ,sizeof(unsigned int),1,finput); 

uint8_t *AcharPix  = new uint8_t[NPix1*NPix2*NPix3];
fread(AcharPix,sizeof(uint8_t),NPix1*NPix2*NPix3,finput);
fclose(finput);

thrust::host_vector<uint8_t> AcharPixB(NPix1*NPix2*NPix3);
for(i = 0; i < NPix1*NPix2*NPix3; i++){
    AcharPixB[i] = AcharPix[i];
} // i

// Copying raw array directly to a device_vector APix
// thrust::device_vector<bool> d_APix(AcharPixB,AcharPixB+NPix1*NPix2*NPix3); 
thrust::device_vector<uint8_t> d_APix=AcharPixB;
res = ((double)vs)/1000.0;
cout << "nx: " << NPix1 << " ny: " << NPix2 << " nz: " << NPix3 << " vs: " << vs << endl;
delete[] AcharPix;

dl         = vstep*res;
// all compartments have equal D for now
dt         = (double) ( Dintra > Dextra ? (dl*dl)/(6.0*Dintra) :  (dl*dl)/(6.0*Dextra) ); 
TN         = ceil(dtime/dt); 
timetemp   = round( ((double)TN)/dtime);
timepoints = (mspoints <= timetemp) ? round(dtime*((double)mspoints) ) : TN;
TN_TP      = ((double)TN)/((double)timepoints);
TN         = ceil( ((double)TN)/((double)timepoints) )*timepoints;

char *initFlagS = (char*)(initFlag == 1 ? "ICS" : ( initFlag == 2 ? "ECS" : ( initFlag == 3 ? "ICS+ECS" : ( initFlag == 4 ? "Center" : "NOT VALID") ) ) );

cout << endl;
cout << "Simulation Parameters (RMS)"      << endl       << endl;
cout << "Random Walkers: "                 << NPar       << endl;
cout << "Voxel Size: "                     << res        << " (um)"    << endl;
cout << "Length Step: "                    << dl         << " (um)"    << endl;
cout << "Time Step: "                      << dt         << " (ms)"    << endl;
cout << "Total Time Simulation: "          << dtime      << " (ms)"    << endl;
cout << "Number of Time Steps: "           << TN         << endl;
cout << "Dintra(t=0): "                    << Dintra     << " um^2/ms" << endl;
cout << "Dextra(t=0): "                    << Dextra     << " um^2/ms" << endl;
cout << "Permeability: "                   << kappa      << " um/ms"   << endl;
cout << "T2intra: "                        << t2intra    << " ms"      << endl;
cout << "T2extra: "                        << t2extra    << " ms"      << endl;
cout << "Simulated Space: "                << initFlagS  << endl;
cout << "Points per mili-second: "         << mspoints   << endl;
cout << "Total Points to Save: "           << timepoints << endl;
cout << "Total Steps/Points: "             << TN_TP      << endl;
cout << endl;

// .......................................................................... //
// ########################################################################## //
// .......................................................................... //

thrust::host_vector<double> Da(Nc);
thrust::host_vector<double> Dr(Nc);
thrust::host_vector<double> T2(Nc);
thrust::host_vector<double> stepDa(Nc); 
thrust::host_vector<double> stepDr(Nc); 
thrust::host_vector<double> Pij(Nc*Nc);
        
Da[0] = 0.0; Da[1] = Dintra; Da[2] = Dextra;
Dr[0] = 0.0; Dr[1] = Dintra; Dr[2] = Dextra;

T2[0] = 1e20/dt;
T2[1] = t2intra/dt;
T2[2] = t2extra/dt;
    
// Step size in IAS in µm (normalized)
for(i = 0; i < Nc; i++){
stepDa[i] = sqrt(6.0*dt*Da[i])/res;
stepDr[i] = sqrt(6.0*dt*Dr[i])/res;
#if DEB == 1 
cout << "step size axial  = " << stepDa[i] << endl;
cout << "step size radial = " << stepDr[i] << endl;
#endif
} // i   

k = 0;
for (i=0; i<Nc; i++){
    for (j=0; j<Nc; j++){
        if ( (i==0) || (j==0) ) {
            Pij[k]=0.0;
        } else if ( i==j ) {
            Pij[k]=1.0;
        } else {
            Pij[k]= kappa * (stepDa[i]*res) / Da[i] * 2/3 / ( 1 + kappa/2*(stepDa[i]*res/Da[i] + stepDa[j]*res/Da[j]) * 2/3 );
//             Pij[k]=fmin(1.0,sqrt(D[j]/D[i]));
        }
        cout<<"permeation probability = i" << i << " j" << j << " Pij" << Pij[k]<<endl;
        k++;
    } // j
} // i
        
thrust::host_vector<double> TD(timepoints);
for (i = 0 ; i < timepoints; i++){
TD[i]=(i*(TN/timepoints)+1)*dt;
}

// ********** Simulate diffusion **********

// Initialize state of RNG
int blockSize = 64;
int numBlocks = (NPar + blockSize - 1) / blockSize;
cout<<numBlocks<<endl<<blockSize<<endl;

thrust::device_vector<hiprandStatePhilox4_32_10_t> devState(numBlocks*blockSize);
setup_kernel<<<numBlocks, blockSize>>>(devState.data().get(),seed);

// Initialize output
thrust::host_vector<double> sig0(timepoints);
thrust::host_vector<double> sig(Nbvec);
thrust::host_vector<double> sigRe(timepoints*Nbvec);
// thrust::host_vector<double> sigIm(timepoints*Nbvec);
thrust::host_vector<double> dx2(timepoints*6);
thrust::host_vector<double> dx4(timepoints*15);
thrust::host_vector<double> NPar_count(timepoints*Nc);
thrust::host_vector<double> NPar_bin(timepoints*Nbin);
for (i=0;i<timepoints;i++){ sig0[i]=0; }
for (i=0;i<Nbvec;i++){ sig[i]=0; }
for (i=0;i<timepoints*Nbvec;i++){ sigRe[i]=0; }
// for (i=0;i<timepoints*Nbvec;i++){ sigIm[i]=0; }    
for (i=0;i<timepoints*6;i++){ dx2[i]=0; }
for (i=0;i<timepoints*15;i++){ dx4[i]=0; }
for (i=0;i<timepoints*Nc;i++){ NPar_count[i]=0; }
for (i=0;i<timepoints*Nbin;i++){ NPar_bin[i]=0; }

// Move data from host to device
thrust::device_vector<double> d_sig0=sig0;
thrust::device_vector<double> d_sig=sig;
thrust::device_vector<double> d_sigRe=sigRe;
// thrust::device_vector<double> d_sigIm=sigIm;
thrust::device_vector<double> d_dx2=dx2;
thrust::device_vector<double> d_dx4=dx4;
thrust::device_vector<double> d_NPar_count = NPar_count;
thrust::device_vector<double> d_NPar_bin = NPar_bin;
thrust::device_vector<double> d_TD=TD;
thrust::device_vector<double> d_stepDa=stepDa;
thrust::device_vector<double> d_stepDr=stepDr;
thrust::device_vector<double> d_T2=T2;
thrust::device_vector<double> d_Pij=Pij;
thrust::device_vector<double> d_grad=grad;
thrust::device_vector<double> d_Delta=Delta;
thrust::device_vector<double> d_delta=delta;
thrust::device_vector<double> d_TE=TE;
thrust::device_vector<double> d_bval=bval;
thrust::device_vector<double> d_bvec=bvec;
// thrust::device_vector<int> d_APix=APix;

//        double *NPar_count; hipMallocManaged(&NPar_count,sizeof(double)); NPar_count[0] = 0;
// Parallel computation
begin=clock();
propagate<<<numBlocks, blockSize>>>(devState.data().get(),
                                    d_sig0.data().get(), d_sig.data().get(), d_sigRe.data().get(),
                                    d_dx2.data().get(), d_dx4.data().get(),
                                    d_NPar_count.data().get(),
                                    d_NPar_bin.data().get(),
                                    d_TD.data().get(), TN,
                                    d_stepDa.data().get(), d_stepDr.data().get(), d_T2.data().get(), d_Pij.data().get(),
                                    Nc, NPar, res, dt,
                                    d_APix.data().get(), NPix1, NPix2, NPix3,
                                    d_bvec.data().get(), d_bval.data().get(), d_grad.data().get(),
                                    d_Delta.data().get(), d_delta.data().get(), d_TE.data().get(),
                                    Nbvec, timepoints, pgse_flag, initFlag);
hipDeviceSynchronize();
end=clock();
cout << "Done! Elpased time "<<double((end-begin)/CLOCKS_PER_SEC) << " s"<< endl;


thrust::copy(d_sig0.begin(), d_sig0.end(), sig0.begin());
thrust::copy(d_sig.begin(), d_sig.end(), sig.begin());
thrust::copy(d_sigRe.begin(), d_sigRe.end(), sigRe.begin());
// thrust::copy(d_sigIm.begin(), d_sigIm.end(), sigIm.begin());
thrust::copy(d_dx2.begin(), d_dx2.end(), dx2.begin());
thrust::copy(d_dx4.begin(), d_dx4.end(), dx4.begin());
thrust::copy(d_NPar_count.begin(), d_NPar_count.end(), NPar_count.begin());
thrust::copy(d_NPar_bin.begin(), d_NPar_bin.end(), NPar_bin.begin());
        
// .......................................................................... //
// ########################################################################## //
// ########################################################################## //
// ########################################################################## //
// .......................................................................... //

if(debug_flag){
// Save number of particles per compartment
char nameDB[500];
sprintf(nameDB,"%s_NPar_count.txt",basename_output);
ofstream fNpout(nameDB);
for(i = 0; i < timepoints; i++){
for(j = 0; j < Nc; j++){
if(j == Nc-1){
fNpout << NPar_count[i*Nc+j] << endl;
}
else{
fNpout<<NPar_count[i*Nc+j]<<"\t";
} // if-else
} // j
} // i
fNpout.close();
} // if debug

// saving PGSE signal
if(pgse_flag){
// copy    
double *SignalPGSE = new double[Nbvec];
for(k = 0; k < Nbvec; k++){ 
SignalPGSE[k] = sig[k];   
// cout << sig[k] << endl ;
}
// saving
// for now it will be saved in a different file  
FILE * pgseFile;      
pgseFile = fopen (namePGSE, "wb");
fwrite (SignalPGSE,sizeof(double),Nbvec,pgseFile);
fclose (pgseFile);
delete[] SignalPGSE;
} // pgse

// Saving output results
// timepoints(1)+TDout(timepoints)+dx2(timepoints*6)+dx4(timepoints*15)+sig0(timepoints)+sig(timepoints)*Nbvec);
unsigned int h, y, x;
unsigned int nmat     = 1;
unsigned int nparams  = 15;
unsigned int ncolumns = 1 + 6 + 15 + 1 + Nbvec;
if ( initFlag == 4 ) { ncolumns += Nbin; }
unsigned int matelem  = ncolumns*timepoints;
unsigned int ntotal   = nmat*ncolumns*timepoints+(nmat*3)+nparams;
double *Out = new double[ntotal];

// Some Parameters (16) of the Simulation
y = 0;
Out[y] = nparams;    y++;  // 01. Number of initialization points (randomly will be 1)
Out[y] = nmat;       y++;  // 02. Number of initialization points (randomly will be 1)
Out[y] = ncolumns;   y++;  // 03. Number of columns of each ini matrix (= 1 x 6 x 15 x Nbvecs)
Out[y] = timepoints; y++;  // 04. Number of time points being saved
Out[y] = dt;         y++;  // 05. Time step in ms
Out[y] = TN;         y++;  // 06. Number of time steps
Out[y] = NPar;       y++;  // 07. Number of random walkers
Out[y] = Nbvec;      y++;  // 08. Number of Diffusion Encoding Gradients
Out[y] = Dintra;     y++;  // 09. Diffusion coefficient inside the axon in um^2/ms
Out[y] = Dextra;     y++;  // 10. Diffusion coefficient outside the axon in um^2/ms
Out[y] = kappa;      y++;  // 11. Permeability of a lipid bi-layer in um/ms
Out[y] = initFlag;   y++;  // 12. Ini pos: 1=ICS, 2=ECS, 3=ICS+ECS+myelin, 4=center
Out[y] = res;        y++;  // 13. Box size
Out[y] = t2intra;    y++;  // 14. T2 inside cell, ms
Out[y] = t2extra;    y++;  // 15. T2 outside cell, ms

// Initialization Points (y is the offset)
for(k = 0; k < nmat; k++){ 
Out[y] = -1; y++;  
Out[y] = -1; y++;  
Out[y] = -1; y++;  
// cout << Out[y-3] << " " << Out[y-2] << " " << Out[y-1] << " " << endl ;
}

// Set outputs to save on a .bin file
for(k = 0; k < nmat; k++){
for(i = 0; i < timepoints; i++){    
// Timepoints
j      = 0;     
x      = y + matelem*k + timepoints*j + i; // (timepoints*j = 0) ToFix: TD should be saved only once
Out[x] = TD[i];
j++;
// sig0   
x      = y + matelem*k + timepoints*j + i; // 
Out[x] = sig0[i];
j++;
// Moment Tensor for dx^2 
for(h = 0; h < 6; h++){
x      = y + matelem*k + timepoints*j + i;     
Out[x] = dx2[k*timepoints*6 + i*6 + h]; 
j++;
} // h
// Moment Tensor for dx^4
for(h = 0; h < 15; h++){
x      = y + matelem*k + timepoints*j + i;     
Out[x] = dx4[k*timepoints*15 + i*15 + h]; 
j++;
} // h
// Signal Re
for(h = 0; h < Nbvec; h++){
x      = y + matelem*k + timepoints*j + i;      
Out[x] = sigRe[k*timepoints*Nbvec + i*Nbvec + h]; 
j++;
} // h

if ( initFlag == 4 ) {
// Particle density in bin
for(h = 0; h < Nbin; h++){
x      = y + matelem*k + timepoints*j + i;
Out[x] = NPar_bin[k*timepoints*Nbin + i*Nbin + h];
j++;
} // h
} // if
        
} // i
} // k

//char nameFile[300];
FILE * outFile;
//sprintf(nameFile,"%s_results.bin",basename_output);
outFile = fopen (basename_output, "wb");
fwrite (Out,sizeof(double),ntotal,outFile);
fclose (outFile);

delete[] Out;

return 0;
} // main


